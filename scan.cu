#include "hip/hip_runtime.h"
// MP 5 Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ... + lst[n-1]}

#include <iterator>
#include <iostream>

#include    <wb.h>

#define BLOCK_SIZE 1024 //@@ You can change this

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

template < typename Operator >
__global__ void scan(float * input, float * output, int len, Operator op )
{
    //@@ Modify the body of this function to complete the functionality of
    //@@ the scan on the device
    //@@ You may need multiple kernel calls; write your kernels before this
    //@@ function and call them from here
   __shared__ float tmp[ BLOCK_SIZE ];

   int t = threadIdx.x;
   int i = blockIdx.x * blockDim.x + t;

   if( i < len )
   {
      tmp[t] = input[i];

      for( unsigned int stride = 1; stride <= t; stride *= 2 )
      {
         __syncthreads();
         float inl = tmp[t-stride];
         __syncthreads();
         tmp[t] = op( tmp[t], inl );
      }

      output[i] = tmp[t];
   }
}

struct Adder
{
   __device__ float operator()( float a, float b ) const { return a+b; }
};

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
    int numElements; // number of elements in the list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numElements);
    hostOutput = (float*) malloc(numElements * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck(hipMalloc((void**)&deviceInput, numElements*sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceOutput, numElements*sizeof(float)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Clearing output memory.");
    wbCheck(hipMemset(deviceOutput, 0, numElements*sizeof(float)));
    wbTime_stop(GPU, "Clearing output memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    wbCheck(hipMemcpy(deviceInput, hostInput, numElements*sizeof(float), hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    dim3 dimGrid( ceil((double)numElements/BLOCK_SIZE), 1, 1 );
    dim3 dimBlock( BLOCK_SIZE, 1, 1 );

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Modify this to complete the functionality of the scan
    //@@ on the deivce
    scan<<< dimGrid, dimBlock >>>( deviceInput
      , deviceOutput
      , numElements
      , Adder() );

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements*sizeof(float), hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    std::copy( hostOutput, hostOutput+numElements
      , std::ostream_iterator<float>(std::cout,"\n") );

    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceInput);
    hipFree(deviceOutput);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    free(hostOutput);

    return 0;
}


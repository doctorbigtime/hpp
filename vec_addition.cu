#include "hip/hip_runtime.h"
// MP 1
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <wb.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
   int i = threadIdx.x + blockDim.x * blockIdx.x;
   printf( "i: %d\n", i );
   if( i < len ) out[i] = in1[i] + in2[i];
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * h_A;
    float * h_B;
    float * h_C;
    float * d_A;
    float * d_B;
    float * d_C;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    h_A = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    h_B = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    size_t size = inputLength*sizeof(float);
    h_C = (float *) malloc(size);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);
    std::cout << "size: " << size << std::endl;
   
   /*for( int i = 0; i < inputLength; ++i ) std::cout << "h_A[" << i << "] = " << h_A[i] << std::endl;
   for( int i = 0; i < inputLength; ++i ) std::cout << "h_B[" << i << "] = " << h_B[i] << std::endl;*/

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    hipError_t err;
    if( hipSuccess != (err = hipMalloc( (void**) &d_A, size )) )
      std::cerr << hipGetErrorString(err) << " on line " << __LINE__ << std::endl;
    if( hipSuccess != (err = hipMalloc( (void**) &d_B, size )) )
      std::cerr << hipGetErrorString(err) << " on line " << __LINE__ << std::endl;
    if( hipSuccess != (err = hipMalloc( (void**) &d_C, size )) )
      std::cerr << hipGetErrorString(err) << " on line " << __LINE__ << std::endl;

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    if( hipSuccess != (err = hipMemcpy( d_A, h_A, size, hipMemcpyHostToDevice )) )
      std::cerr << hipGetErrorString(err) << " on line " << __LINE__ << std::endl;
    if( hipSuccess != (err = hipMemcpy( d_B, h_B, size, hipMemcpyHostToDevice )) )
      std::cerr << hipGetErrorString(err) << " on line " << __LINE__ << std::endl;

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
    dim3 dimGrid( ceil(inputLength/256.0), 1, 1 );
    dim3 dimBlock( 256, 1, 1 );
    
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    vecAdd<<< dimGrid, dimBlock >>>(d_A,d_B,d_C,inputLength);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    if( hipSuccess != (err = hipMemcpy( h_C, d_C, size, hipMemcpyDeviceToHost )) )
      std::cerr << hipGetErrorString(err) << " on line " << __LINE__ << std::endl;
    for( int i = 0; i < inputLength; ++i ) std::cout << "h_C[" << i << "] = " << h_C[i] << std::endl;

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    if( hipSuccess != (err = hipFree(d_A)) )
      std::cerr << hipGetErrorString(err) << " on line " << __LINE__ << std::endl;
    if( hipSuccess != (err = hipFree(d_B)) )
      std::cerr << hipGetErrorString(err) << " on line " << __LINE__ << std::endl;
    if( hipSuccess != (err = hipFree(d_C)) )
      std::cerr << hipGetErrorString(err) << " on line " << __LINE__ << std::endl;

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, h_C, inputLength);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
